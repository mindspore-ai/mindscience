/**
 * Copyright 2023 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipfft/hipfft.h>
#include "ifft_3d.cuh"

extern "C"
int IFFT3D(int nparam, void **params, int *ndims, int64_t **shapes, const char **dtypes, void *stream, void *extra) {
    if (nparam != 2 || ndims[0] != 3) {
        return 1;
    }
    if (strcmp(dtypes[0], "complex64") != 0) {
        return 2;
    }

    hipStream_t custream = static_cast<hipStream_t>(stream);
    void *input = params[0];
    void *output = params[1];

    int fftx = shapes[0][0];
    int ffty = shapes[0][1];
    int fftz = shapes[0][2];

    hipfftHandle FFT_plan_c2r;
    hipfftPlan3d(&FFT_plan_c2r, fftx, ffty, (fftz - 1) * 2, HIPFFT_C2R);
    hipfftSetStream(FFT_plan_c2r, custream);
    hipfftExecC2R(FFT_plan_c2r, static_cast<hipfftComplex *>(input), static_cast<hipfftReal *>(output));

    return 0;
}
